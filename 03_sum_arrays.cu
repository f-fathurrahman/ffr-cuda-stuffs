
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <time.h>

void sum_arrays_on_host( const int N, double *A, double *B, double *C )
{
    for( int idx = 0; idx < N; idx++ ) {
        C[idx] = A[idx] + B[idx];
    }
}

void initial_data( int Ndata, double *dat )
{
    // Generate different seed for random number
    time_t t;
    srand( (unsigned int) time(&t) );

    for( int i = 0; i < Ndata; i++ ) {
        dat[i] = (double)( rand() & 0xFF )/10.0f;
    }
}

void print_data( int Ndata, double *A )
{
    for( int i = 0; i < Ndata; i++ ) {
        printf("%8d %18.10f\n", i, A[i]);
    }
}

int main( int argc, char **argv)
{
    int Ndata = 10;
    size_t Nbytes = Ndata * sizeof(double);

    double *h_A, *h_B, *h_C;

    h_A = (double*) malloc(Nbytes);
    h_B = (double*) malloc(Nbytes);
    h_C = (double*) malloc(Nbytes);

    initial_data( Ndata, h_A );
    initial_data( Ndata, h_B );

    sum_arrays_on_host( Ndata, h_A, h_B, h_C );

    for( int i = 0; i < Ndata; i++ ) {
        printf("%3d %18.10f %18.10f %18.10f\n", i, h_A[i], h_B[i], h_C[i]);
    }

    return 0;
}
