#include "hip/hip_runtime.h"
#include "kernel.h"
#include <cstdio>
#define TPB 32

__device__ float distance(float x1, float x2)
{
    return sqrt( (x2 - x1)*(x2 - x1) );
}

__global__ void distanceKernel(float *d_out, float *d_in, float ref)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    const float x = d_in[i];

    d_out[i] = distance(x, ref);

    printf("%2d %18.10f %18.10f %18.10f\n", i, ref, x, d_out[i]);
}

void distanceArray(float *out, float *in, float ref, int len)
{
    float *d_in = 0;
    float *d_out = 0;

    hipMalloc(&d_in, len*sizeof(float));
    hipMalloc(&d_out, len*sizeof(float));

    // copy input data from host to device
    hipMemcpy(d_in, in, len*sizeof(float), hipMemcpyHostToDevice);

    // launch the kernel
    distanceKernel<<<len/TPB, TPB>>>(d_out, d_in, ref);

    // copy the result
    hipMemcpy(out, d_out, len*sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(d_in);
    hipFree(d_out);
}
