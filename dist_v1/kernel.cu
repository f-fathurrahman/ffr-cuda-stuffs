
#include <hip/hip_runtime.h>
#include <cstdio>
#define N 64
#define TPB 16

__device__ float scale(int i, int n)
{
    return ((float) i)/(n - 1);
}

__device__ float distance(float x1, float x2)
{
    return sqrt( (x2 - x1)*(x2 - x1) );
}

__global__ void distanceKernel(float *d_out, float ref, int len)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;

    // calculate i-th point
    const float x = scale(i, len);

    // Calculate distance
    d_out[i] = distance(x, ref);

    printf("%2d %f %f %f\n", i, ref, x, d_out[i]);
}

int main()
{
    const float ref = 0.5f;

    float *d_out = 0;
    hipMalloc( &d_out, N*sizeof(float) );

    distanceKernel<<<N/TPB, TPB>>>(d_out, ref, N);

    hipFree(d_out);

    return 0;
}