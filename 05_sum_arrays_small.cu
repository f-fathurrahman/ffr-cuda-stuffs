#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

void sum_arrays_on_host( const int N, float *A, float *B, float *C )
{
    for( int idx = 0; idx < N; idx++ ) {
        C[idx] = A[idx] + B[idx];
    }
}

void initial_data( int Ndata, float *dat )
{
    // Generate different seed for random number
    time_t t;
    srand( (unsigned int) time(&t) );

    for( int i = 0; i < Ndata; i++ ) {
        dat[i] = (float)( rand() & 0xFF )/10.0f;
    }
}

__global__ void sum_arrays_on_gpu( float *A, float *B, float *C )
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

int check_result( const int N, float *hostRef, float *gpuRef )
{
    double epsilon = 1e-8;
    for( int i = 0; i < N; i++ ) {
        if( abs(hostRef[i] - gpuRef[i]) > epsilon ) {
            printf("Arrays do not match!\n");
            printf("i = %3d , host = %5.2f , device = %5.2f\n", i, hostRef[i], gpuRef[i]);
            return -1;
        }
    }
    return 0;
}


int main( int argc, char** argv )
{
    printf("%s starting ...\n", argv[0]);

    // Set up device
    int dev = 0;
    hipSetDevice(dev);

    int Ndata = 33;

    // host memory
    size_t Nbytes = Ndata*sizeof(float);

    float *h_A, *h_B, *hostRef, *gpuRef;

    h_A     = (float*) malloc(Nbytes); 
    h_B     = (float*) malloc(Nbytes); 
    hostRef = (float*) malloc(Nbytes);
    gpuRef  = (float*) malloc(Nbytes);

    // Initialize data at host side
    initial_data( Ndata, h_A );
    initial_data( Ndata, h_B );

    memset( hostRef, 0, Nbytes );
    memset( gpuRef, 0, Nbytes );

    // Initialize device memory
    float *d_A, *d_B, *d_C;
    hipMalloc( (float**)&d_A, Nbytes );
    hipMalloc( (float**)&d_B, Nbytes );
    hipMalloc( (float**)&d_C, Nbytes );

    // transfer data
    hipMemcpy( d_A, h_A, Nbytes, hipMemcpyHostToDevice );
    hipMemcpy( d_B, h_B, Nbytes, hipMemcpyHostToDevice );

    // invoke kernel
    dim3 block(Ndata);
    dim3 grid(Ndata/block.x);

    sum_arrays_on_gpu<<<grid,block>>>( d_A, d_B, d_C );

    printf("Execution configuration: <<<%d,%d>>>\n", grid.x, block.x);

    // 
    sum_arrays_on_host( Ndata, h_A, h_B, hostRef );

    hipMemcpy( gpuRef, d_C, Nbytes, hipMemcpyDeviceToHost );

    int err = check_result( Ndata, hostRef, gpuRef );
    if( err == 0 ) {
        printf("Test passed\n");
    }

    hipFree( d_A );
    hipFree( d_B );
    hipFree( d_B );

    hipDeviceReset();

    return 0;
}