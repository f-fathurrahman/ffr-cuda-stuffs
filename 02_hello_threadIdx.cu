
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void hello_from_gpu()
{
  printf("This is hello from GPU threadIdx.x = %d\n", threadIdx.x);
}

int main()
{
  printf("This is hello from CPU\n");

  hello_from_gpu <<<1,10>>> ();

  hipDeviceReset();

  return 0;
}
